#include "hip/hip_runtime.h"
#include <cstdlib>
#include <cstring>
#include <iostream>
#include "mergesort.h"

// Device kernel function
__global__ void vector_mul(int *a_dev, int *b_dev, int *c_dev){
    //Element-wise vector multiplication
    unsigned idx = blockIdx.x * blockDim.x + threadIdx.x;
    c_dev[idx] = a_dev[idx] + b_dev[idx];
}
// Perform a full mergesort on our section of the data.
//
__global__ void gpu_mergesort(long* source, long* dest, long size, long width, long slices, dim3* threads, dim3* blocks) {
    unsigned idx = blockIdx.x * blockDim.x + threadIdx.x;
    long start = width*idx*slices, 
         middle, 
         end;

    for (long slice = 0; slice < slices; slice++) {
        if (start >= size)
            break;

        middle = min(start + (width / 2), size);
        end = min(start + width, size);
        gpu_bottomUpMerge(source, dest, start, middle, end);
        start += width;
    }
}

//
// Finally, sort something
// gets called by gpu_mergesort() for each slice
//
__device__ void gpu_bottomUpMerge(long* source, long* dest, long start, long middle, long end) {
    long i = start;
    long j = middle;
    for (long k = start; k < end; k++) {
        if (i < middle && (j >= end || source[i] < source[j])) {
            dest[k] = source[i];
            i++;
        } else {
            dest[k] = source[j];
            j++;
        }
    }
}

void mergesort(unsigned *m_data) {
    /* Assignment */

    using namespace std;   
    // get vector vector_size
    unsigned vector_size = num_data;


    // Host code
    int *a = 0, *b = 0, *c = 0;
    int *a_dev = 0, *b_dev = 0, *c_dev = 0;

    // Host data
    a = (int*)malloc(vector_size*sizeof(int));
    b = (int*)malloc(vector_size*sizeof(int));
    c = (int*)malloc(vector_size*sizeof(int));

    for(unsigned i = 0; i < vector_size; i++){
        a[i] = m_data[i];
        b[i] = m_data[i];
    }
    memset(c,0,vector_size*sizeof(int));

    // Device memory allocation
    hipMalloc(&a_dev, vector_size * sizeof(int));
    hipMalloc(&b_dev, vector_size * sizeof(int));
    hipMalloc(&c_dev, vector_size * sizeof(int));

    // Memory copy from host to device memory
    hipMemcpy(a_dev, a, vector_size * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(b_dev, b, vector_size * sizeof(int), hipMemcpyHostToDevice);

    // Kernel launch
    //vector_mul<<<vector_size/block_size,block_size>>>(a_dev,b_dev,c_dev);

    // Actually call the kernel
    //gpu_mergesort<<<blocksPerGrid, threadsPerBlock>>>(A, B, size, width, slices, D_threads, D_blocks);

    for (int width = 2; width < num_data*2; width *= 2) {
        long slices = num_data / (block_size * width) + 1;

        // Actually call the kernel
        gpu_mergesort<<<blocksPerGrid, threadsPerBlock>>>(a_dev, c_dev, num_data, width, slices);

        // memory copy from device to host memory
        hipMemcpy(c,c_dev,vector_size * sizeof(int), hipMemcpyDeviceToHost);

        for(unsigned i = 0; i < vector_size; i++){
            a[i] = c[i];
        }
        hipMemcpy(a_dev, a, vector_size * sizeof(int), hipMemcpyHostToDevice);
    }

    // memory copy from device to host memory
    hipMemcpy(c,c_dev,vector_size * sizeof(int), hipMemcpyDeviceToHost);

   
    
    // print vector add results
    cout << "a = [";
    for(unsigned i=0; i<10; i++) {cout << a[i] << " "; }
    cout << "\b]" << endl;
    cout << "*" << endl;
    cout << "b = [";
    for(unsigned i=0; i<10; i++) {cout << b[i] << " "; }
    cout << "\b]" << endl;
    cout << "=" << endl;
    cout << "c = [";
    for(unsigned i=0; i<10; i++) {cout << c[i] << " "; }
    cout << "\b]" << endl;
    

    // Host memory deallocation
    free(a);free(b);free(c);

    // Device memory deallocation
    hipFree(a_dev); hipFree(b_dev); hipFree(c_dev);

}

